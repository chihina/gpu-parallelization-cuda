//     Author: Chihiro Nakatani
//     February 9th, 2021
//     This script contains the inner product function with cpp.


#include <hip/hip_runtime.h>
#include <iostream>

// Define kernel function for inner product
__global__
void inner_product(int n, float *x, float *y, float *z)
{
    for (int i = 0; i < n; i++)
        z[0] += x[i] * y[i];
}

// Define main function
int main(void)
{
    //   Define input vector length
    int N = 5;
    std::cout << "Vector size : " << N << std::endl;  
   
    //   Initialize a float variable
    float *z = new float[1];
    hipMallocManaged(&z, 1*sizeof(float));
    z[0] = 0;

    // Initialize float vectors with different float values
    float *x = new float[N];
    float *y = new float[N];
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Check whether Initialization is right (If you use big N, you should not check it in command line)
    std::cout << "Initialzze scalar z: " << z[0] << std::endl;  

    std::cout << "Initialize vector x: [ ";
    for (int i = 0; i < N; i++) {
        std::cout << x[i] << " ";
    }
    std::cout << "]" << std::endl;

    std::cout << "Initialize vector y: [ ";
    for (int i = 0; i < N; i++) {
        std::cout << y[i] << " ";
    }
    std::cout << "]" << std::endl;

    // Execute kernel on vector on the GPU
    inner_product<<<1, 1>>>(N, x, y, z);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    std::cout << "Inner product (z = (x,y)): " << z[0] << std::endl;

    // Free memory which is used for vectors
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}