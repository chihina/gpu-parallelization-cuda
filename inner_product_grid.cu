//     Author: Chihiro Nakatani
//     February 9th, 2021
//     This script contains the inner product function with cpp.


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#define SIZE_OF_ARRAY(array) (sizeof(array)/sizeof(array[0]))

// Define kernel function for inner product
__global__
void inner_product(int n, float *x, float *y, float *z)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        z[i] = x[i] * y[i];
}

// Define main function
int main(void)
{
    //   Define input vector length
    int N = 10000000;
    std::cout << "Vector size : " << N << std::endl;  
   
    // Initialize float vectors with different float values
    float *x = new float[N];
    float *y = new float[N];
    float *z = new float[N];
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    hipMallocManaged(&z, N*sizeof(float));
    for (int i = 0; i < N; i++) {
        x[i] = 1;
        y[i] = 2;
        z[i] = 0;
    }

    // Check whether Initialization is right (If you use big N, you should not check it in command line)

    //std::cout << "Initialize vector x: [ ";
    //for (int i = 0; i < N; i++) {
    //    std::cout << x[i] << " ";
    //}
    //std::cout << "]" << std::endl;

    //std::cout << "Initialize vector y: [ ";
    //for (int i = 0; i < N; i++) {
    //    std::cout << y[i] << " ";
    //}
    //std::cout << "]" << std::endl;

    // Execute kernel on vector on the GPU
    int blockSize = 256;
    int numBlocks = (N+blockSize-1) / blockSize;    
    inner_product<<<numBlocks, blockSize>>>(N, x, y, z);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    float inner_product_value = std::accumulate(z, z + N, 0);
    std::cout << "Inner product (z = (x,y)): " << inner_product_value << std::endl;

    // Free memory which is used for vectors
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}